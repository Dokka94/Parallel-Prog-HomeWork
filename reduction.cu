#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <string.h>

#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// CUDA kernel. Each thread takes care of one element of c
__global__ void first(float *c,float *d, int n)
{
    // Get our global thread ID
    int tid = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    __shared__ float temp[1024];
    if (tid < n)
      temp[threadIdx.x] = c[tid];
    else
      temp[threadIdx.x] = 0;
    for(int j=blockDim.x>>1;j>=1;j>>=1){
      __syncthreads();
      if (threadIdx.x < j) temp[threadIdx.x] += temp[threadIdx.x+j];
    }
    //float sum = 0;
    //for(int i=0; i<1024;i++) sum += tmp[i];
    if (threadIdx.x == 0) atomicAdd(d,temp[0]);
    //if (threadIdx.x == 0) d[blockIdx.x] = temp[0];
} 
 
int main( int argc, char* argv[] )
{
    // Size of vectors
    int n = 2000;
 
    //Host vector
    float *h_c;
 
    //Device output vector
    float *d_c;
    float *d_d;
        
    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(float);
 
    // Allocate memory on host
    h_c = (float*)malloc(bytes);

    for(int i=0;i<n;i++)
    h_c[i] = 1;
 
    // Allocate memory on GPU
    cudaCheck(hipMalloc(&d_c, bytes));
    cudaCheck(hipMalloc(&d_d, sizeof(float)));
    cudaCheck(hipMemset(d_d,0.0, sizeof(float)));
    cudaCheck(hipMemcpy(d_c,h_c,bytes, hipMemcpyHostToDevice));
 
    // Copy host vectors to device
    int blockSize, gridSize;

    // Number of threads in each thread block
    blockSize = 1024;
 
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);
 
    // Execute the kernel
    double t1 = omp_get_wtime();
    first<<<gridSize, blockSize>>>(d_c, d_d, n);
 
    // Synchronize
    cudaCheck(hipDeviceSynchronize());
    double elapsed = omp_get_wtime()-t1;
    printf("Time: %g\n", elapsed);

    // Copy array back to host
    cudaCheck(hipMemcpy( h_c, d_d, sizeof(float), hipMemcpyDeviceToHost));
    printf("Checksum: %f\n",h_c[0]);
    // Release device memory
    hipFree(d_c);
 
    // Release host memory
    free(h_c);
 
    return 0;
}

